/*
 *  CNC: Concurrent Number Cruncher
 *  Copyright (C) 2008 GOCAD/ASGA, INRIA/ALICE
 *
 *  This program is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation; either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program; if not, write to the Free Software
 *  Foundation, Inc., 675 Mass Ave, Cambridge, MA 02139, USA.
 *
 *  If you modify this software, you should include a notice giving the
 *  name of the person performing the modification, the date of modification,
 *  and the reason for such modification.
 *
 *  Contact: Luc Buatois
 *
 *     buatois@gocad.org
 *
 *     ASGA-INPL Bt. G
 *     Rue du Doyen Marcel Roubault - BP 40
 *     54501 VANDOEUVRE LES NANCY
 *     FRANCE
 *
 *  Note that the GNU General Public License does not permit incorporating
 *  the Software into proprietary programs. 
 */

// Header files

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

// Thread block size (THREAD_BLOCK_SIZE� = 768 at max)
#define THREAD_BLOCK_SIZE 16

//---------------------------------------------------------------------------//

dim3 CNCdimGrid_vec  ;
dim3 CNCdimBlock_vec ;
dim3 CNCdimGrid_vec2 ;
dim3 CNCdimBlock_vec2;
dim3 CNCdimGrid_vec4 ;
dim3 CNCdimBlock_vec4;

//---------------------------------------------------------------------------//

extern "C" void cnc_cuda_set_dim_vec ( unsigned int dim_grid_x, unsigned int dim_grid_y,
								   unsigned int dim_block_x, unsigned int dim_block_y ) ;

extern "C" void cnc_cuda_set_dim_vec2( unsigned int dim_grid_x, unsigned int dim_grid_y,
								   unsigned int dim_block_x, unsigned int dim_block_y ) ;

extern "C" void cnc_cuda_set_dim_vec4( unsigned int dim_grid_x, unsigned int dim_grid_y,
								   unsigned int dim_block_x, unsigned int dim_block_y ) ;

extern "C" void cnc_cuda_vecvecmult( unsigned int size, float * x, float * y, float * r ) ;

extern "C" void cnc_cuda_mat1x1vecmult ( float * matrix, unsigned int size_matrix,
									 uint2 * rowptr, unsigned int size_rowptr,
									 unsigned int * colind, unsigned int size_colind,
									 float * x, float * b, unsigned int size_vec ) ;

extern "C" void cnc_cuda_mat2x2vecmult4 ( float4 * matrix, unsigned int size_matrix,
									  uint2 * rowptr, unsigned int size_rowptr,
									  unsigned int * colind, unsigned int size_colind,
									  float2 * x, float2 * b, unsigned int size_vec ) ;

extern "C" void cnc_cuda_mat4x4vecmult4 ( float4 * mat0,  float4 * mat1,
									  float4 * mat2,  float4 * mat3,
									  unsigned int size_matrix,
									  uint2 * rowptr, unsigned int size_rowptr,
									  unsigned int * colind, unsigned int size_colind,
									  float4 * x, float4 * b, unsigned int size_vec ) ;

//---------------------------------------------------------------------------//
// Forward declaration of the blas kernels									 //
//---------------------------------------------------------------------------//

__global__ void CNCVecVecMultKernel ( unsigned int size, float * x, float * y, float * r ) ;

__global__ void CNCMat1x1VecMultKernel ( float * matrix, unsigned int size_matrix,
									  uint2 * rowptr, unsigned int size_rowptr,
									  unsigned int * colind, unsigned int size_colind,
									  float * x, float * b, unsigned int size_vec ) ;

__global__ void CNCMat2x2VecMult4Kernel ( float4 * matrix, unsigned int size_matrix,
									  uint2 * rowptr, unsigned int size_rowptr,
									  unsigned int * colind, unsigned int size_colind,
									  float2 * x, float2 * b, unsigned int size_vec ) ;

__global__ void CNCMat4x4VecMult4Kernel ( float4 * mat0, float4 * mat1,
									   float4 * mat2, float4 * mat3,
									   unsigned int size_matrix,
									   uint2  * rowptr, unsigned int size_rowptr,
									   unsigned int * colind, unsigned int size_colind,
									   float4 * x, float4 * b, unsigned int size_vec ) ;

//---------------------------------------------------------------------------//
// Setup functions and call of the kernels									 //
//---------------------------------------------------------------------------//

void cnc_cuda_set_dim_vec ( unsigned int dim_grid_x, unsigned int dim_grid_y,
					    unsigned int dim_block_x, unsigned int dim_block_y ) {
	CNCdimGrid_vec.x  = dim_grid_x ;
	CNCdimGrid_vec.y  = dim_grid_y ;
	CNCdimBlock_vec.x = dim_block_x ;
	CNCdimBlock_vec.y = dim_block_y ;
}
void cnc_cuda_set_dim_vec2 ( unsigned int dim_grid_x, unsigned int dim_grid_y,
						 unsigned int dim_block_x, unsigned int dim_block_y ) {
	CNCdimGrid_vec2.x  = dim_grid_x ;
	CNCdimGrid_vec2.y  = dim_grid_y ;
	CNCdimBlock_vec2.x = dim_block_x ;
	CNCdimBlock_vec2.y = dim_block_y ;
}
void cnc_cuda_set_dim_vec4 ( unsigned int dim_grid_x, unsigned int dim_grid_y,
						 unsigned int dim_block_x, unsigned int dim_block_y ) {
	CNCdimGrid_vec4.x  = dim_grid_x ;
	CNCdimGrid_vec4.y  = dim_grid_y ;
	CNCdimBlock_vec4.x = dim_block_x ;
	CNCdimBlock_vec4.y = dim_block_y ;
}

//---------------------------------------------------------------------------//

void cnc_cuda_mat1x1vecmult ( float * matrix, unsigned int size_matrix,
						  uint2 * rowptr, unsigned int size_rowptr,
						  unsigned int * colind, unsigned int size_colind,
						  float * x, float * b, unsigned int size_vec ) {

	// Launch the device computation
	CNCMat1x1VecMultKernel<<<CNCdimGrid_vec, CNCdimBlock_vec>>>(
		matrix, size_matrix, rowptr, size_rowptr,
		colind, size_colind, x, b, size_vec);
}

//---------------------------------------------------------------------------//

void cnc_cuda_mat2x2vecmult4 ( float4 * matrix, unsigned int size_matrix,
						   uint2 * rowptr, unsigned int size_rowptr,
						   unsigned int * colind, unsigned int size_colind,
						   float2 * x, float2 * b, unsigned int size_vec ) {

	// Launch the device computation
	CNCMat2x2VecMult4Kernel<<<CNCdimGrid_vec2, CNCdimBlock_vec2>>>(
		matrix, size_matrix,
		rowptr, size_rowptr,
		colind, size_colind,
		x, b, size_vec);
}

//---------------------------------------------------------------------------//

void cnc_cuda_mat4x4vecmult4 ( float4 * mat0,  float4 * mat1,
						   float4 * mat2,  float4 * mat3,
						   unsigned int size_matrix,
						   uint2 * rowptr, unsigned int size_rowptr,
						   unsigned int * colind, unsigned int size_colind,
						   float4 * x, float4 * b, unsigned int size_vec ) {

    // Launch the device computation
	CNCMat4x4VecMult4Kernel<<<CNCdimGrid_vec4, CNCdimBlock_vec4>>>(
												mat0, mat1, mat2, mat3,
											    size_matrix,
											    rowptr, size_rowptr,
											    colind, size_colind,
											    x, b, size_vec);
}

//---------------------------------------------------------------------------//

void cnc_cuda_vecvecmult( unsigned int size, float * x, float * y, float * r ) {

    // Launch the device computation
    CNCVecVecMultKernel<<<CNCdimGrid_vec, CNCdimBlock_vec>>>(size, x, y, r);
}

//---------------------------------------------------------------------------//

__device__ unsigned int compute_thread_index () {

	return ( blockIdx.x*THREAD_BLOCK_SIZE*THREAD_BLOCK_SIZE+
			 blockIdx.y*THREAD_BLOCK_SIZE*THREAD_BLOCK_SIZE*gridDim.x+
			 threadIdx.x+threadIdx.y*THREAD_BLOCK_SIZE) ;
}

//---------------------------------------------------------------------------//

__global__ void CNCVecVecMultKernel ( unsigned int size,
								   float * x,
								   float * y,
								   float * r ) {

	// Thread index
	const unsigned int index = compute_thread_index () ;
		
	if ( index < size )
		r[index] = x[index]*y[index] ;
}

//---------------------------------------------------------------------------//

__global__ void CNCMat1x1VecMultKernel ( float * matrix, unsigned int size_matrix,
									  uint2 * rowptr, unsigned int size_rowptr,
									  unsigned int * colind, unsigned int size_colind,
									  float * x, float * b, unsigned int size_vec ) {


	// Thread index
	const unsigned int index = compute_thread_index () ;

	if ( index < size_vec ) {

		uint2 rowptr_bounds = rowptr[index] ;

		float res = 0.0f ;

		// for each block of the block_row, mult
		for ( unsigned int i=rowptr_bounds.x; i<rowptr_bounds.y; i++ ) { 
			res += matrix[i]*x[colind[i]] ;
		}
		b[index] = res ;
	}
}

//---------------------------------------------------------------------------//

__global__ void CNCMat2x2VecMult4Kernel ( float4 * matrix, unsigned int size_matrix,
									  uint2 * rowptr, unsigned int size_rowptr,
									  unsigned int * colind, unsigned int size_colind,
									  float2 * x, float2 * b, unsigned int size_vec ) {

	// Thread index
	const unsigned int index = compute_thread_index () ;

	if ( index<<1 < size_vec ) {

		uint2 rowptr_bounds = rowptr[index] ;
		float2 res ;
		res.x = res.y = 0.0f ;

		unsigned int ci = 0 ;
		float2 x_vec ;
		float4 mat_vec ;

		// for each block of the block_row, mult
		for ( int i=rowptr_bounds.x; i<rowptr_bounds.y; i++ ) { 

			ci = colind[i] ;
			mat_vec = matrix[i] ;
			x_vec = x[ci] ;

			res.x += mat_vec.x*x_vec.x+mat_vec.y*x_vec.y ;
			res.y += mat_vec.z*x_vec.x+mat_vec.w*x_vec.y ;
		}
		b[index] = res ;
	}
}

//---------------------------------------------------------------------------//

__global__ void CNCMat4x4VecMult4Kernel ( float4 * mat0, float4 * mat1,
									   float4 * mat2, float4 * mat3,
									   unsigned int size_matrix,
									   uint2 * rowptr, unsigned int size_rowptr,
									   unsigned int * colind, unsigned int size_colind,
									   float4 * x, float4 * b, unsigned int size_vec ) {

    // Thread index
	const unsigned int index = compute_thread_index () ;

	if ( index<<2 < size_vec ) {

		uint2 rowptr_bounds = rowptr[index] ;
		float4 res ;
		res.x=res.y=res.z=res.w = 0.0f ;

		unsigned int ci = 0 ;
		float4 x_vec ;
		float4 row0  ;
		float4 row1  ;
		float4 row2  ;
		float4 row3  ;

		// for each block of the block_row, mult
		for ( int i=rowptr_bounds.x; i<rowptr_bounds.y; i++ ) {
			row0 = mat0[i] ;
			row1 = mat1[i] ;
			row2 = mat2[i] ;
			row3 = mat3[i] ;
			ci = colind[i] ;
			x_vec = x[ci]  ;
			res.x += row0.x*x_vec.x+row0.y*x_vec.y+row0.z*x_vec.z+row0.w*x_vec.w ;
			res.y += row1.x*x_vec.x+row1.y*x_vec.y+row1.z*x_vec.z+row1.w*x_vec.w ;
			res.z += row2.x*x_vec.x+row2.y*x_vec.y+row2.z*x_vec.z+row2.w*x_vec.w ;
			res.w += row3.x*x_vec.x+row3.y*x_vec.y+row3.z*x_vec.z+row3.w*x_vec.w ;
		}
		b[index] = res ;
	}
}

//---------------------------------------------------------------------------//
